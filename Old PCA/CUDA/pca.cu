#include "hip/hip_runtime.h"
/*==================================================================================================
 *	pca.cu
 *
 *  Edited by: William Halsey
 *  whalsey@g.clemson.edu
 *
 *  THIS FILE CONTAINS
 *      calculate_projected_images
 *      main
 *
 *	Description: 
 *
 *  Last edited: Jul. 18, 2013
 *  Edits: 
 *
 */
#ifdef WIN32
#include <windows.h>
#endif
#include "pca.h"
#include <time.h>

/*==================================================================================================
 *	calculate_projected_images
 *
 *	parameters
 *      double pointer, type eigen_type = projectedtrainimages
 *      double pointer, type eigen_type = projectedimages
 *      double pointer, type eigen_type = eigenfacesT
 *      single pointer, type long int   = images
 *      single pointer, type long int   = imgsize
 *      single pointer, type long int   = facessize
 *
 *	returns
 *      N/A
 *
 *	Description: 
 *
 *  THIS FUNCTION CALLS
 *
 *  THIS FUNCTION IS CALLED BY
 *      main    (pca.cu)
 *
 */
void calculate_projected_images(eigen_type **projectedtrainimages, eigen_type **projectedimages,
    eigen_type **eigenfacesT, long int *images, long int *imgsize, long int *facessize) {

	eigen_type *projectedtrainimages_d, *projectedimages_d, *eigenfacesT_d;
	long int i;

	/*  allocate result matrix (no, you HAVE to do this... just try NOT doing it.)  */
	(*projectedtrainimages) = (eigen_type *)malloc(sizeof(eigen_type*) * (*images) * (*facessize));
	for(i = 0; i < (*images) * (*facessize); i++)
		(*projectedtrainimages)[i] = 0; //  rand();

 	/*  Allocate device memory for the matrices */
	cublasAlloc((*images) * (*imgsize), sizeof(eigen_type), (void**)&projectedimages_d);
	cublasAlloc((*facessize) * (*imgsize), sizeof(eigen_type), (void**)&eigenfacesT_d);
	cublasAlloc((*images) * (*facessize), sizeof(eigen_type), (void**)&projectedtrainimages_d);
	
	/*  Initialize the device matrices with the host matrices   */
	hipblasSetVector((*images) * (*imgsize), sizeof(eigen_type), (*projectedimages), 1, projectedimages_d, 1);
	hipblasSetVector((*facessize) * (*imgsize), sizeof(eigen_type), (*eigenfacesT), 1, eigenfacesT_d, 1);
	hipblasSetVector((*images) * (*facessize), sizeof(eigen_type), (*projectedtrainimages), 1, projectedtrainimages_d, 1);
	hipDeviceSynchronize();
  	
	/*  Performs operation using cublas */
	hipblasSgemm('n', 't', (*images), (*facessize), (*imgsize), 1, projectedimages_d, (*images), eigenfacesT_d, (*facessize), 1, projectedtrainimages_d, (*images));
	hipDeviceSynchronize();
	
	if(cublasGetError() != HIPBLAS_STATUS_SUCCESS) {
		printf("there was a problem using CUDA/CUBLAS...check your setup!\n");
	}

	/*  Read the result back    */
	hipblasGetVector((*images) * (*facessize), sizeof(eigen_type), projectedtrainimages_d, 1, (*projectedtrainimages), 1);
	
	hipDeviceSynchronize();
	cublasFree(projectedtrainimages_d);
	cublasFree(eigenfacesT_d);
	cublasFree(projectedimages_d);
	
	return;
}


/*==================================================================================================
 *	main
 *
 *	Description: 
 *
 *  THIS FUNCTION CALLS
 *      LoadTrainingDatabase        (matrix_ops.cu)
 *      calculate_projected_images  (pca.cu)
 *      cudasafe                    (pca_host.cu)
 *      Recognition                 (pca_host.cu)
 *
 */
int main(int argc, char *argv[]) {
	double totalTime, itime;
	srand(10);
	long int images, imgsize, facessize, i;
	eigen_type  *projectedimages, *eigenfacesT; /*  2D matrices for eigenfaces and projected images read in from MATLAB file    */
	eigen_type  *projectedtrainimages;  /*  calculated from the read in MATLAB file */
	eigen_type *mean;   /*  1D matrix of average pixel values from training database also read in from MATLAB file  */

	eigen_type *database_d, *image_d, *mean_d, *eigenfacesT_d, *test_image_norm;    /*  pointers to device memory; a.k.a. GPU   */
	Pixel *test_image_d;
	eigen_type *test_image_d2;
	int *recognized_index_d;
	char errormsg[80];

//	for(i = 0; i < 20; i++) {
//		int j = 1 + (int)( 100.0 * rand() / ( RAND_MAX + 1.0 ) );
//		printf("%d, ", j);
//	}
	
	/*  test image numbers present in PCA directory (ppm image files)   */
//	printf("%s", argv[1]);
	char inputimage[30];
	totalTime = -(double)(clock())/CLOCKS_PER_SEC;
	
	
	if(argc != 2) {
		printf("Invalid arguments.\n\t./pca TrainingDBFile\n");
		return 0;
	}
	itime = -(double)(clock())/CLOCKS_PER_SEC;
	LoadTrainingDatabase(argv[1], &projectedimages, &eigenfacesT, &mean, &images, &imgsize, &facessize);
	printf("LoadTrainingDatabase time: %f\n", itime + (double)(clock())/CLOCKS_PER_SEC);
	/*  init CUDA library that is primarily used for matrices   */
	
	itime = -(double)(clock())/CLOCKS_PER_SEC;
	cublasInit();
	
	calculate_projected_images(&projectedtrainimages, &projectedimages, &eigenfacesT, &images, &imgsize, &facessize);
	printf("Calculate_projected_images time: %f\n", itime + (double)(clock())/CLOCKS_PER_SEC);
	itime = -(double)(clock())/CLOCKS_PER_SEC;
	/*  allocate arrays on device   */
	
	sprintf(errormsg, "Failed to allocate the image database on the CUDA device!");
	cudasafe(
		hipMalloc((void **)&database_d,images*facessize*sizeof(eigen_type)), 
		errormsg);
	sprintf(errormsg, "Failed to allocate test image on the CUDA device!");
	cudasafe(
		hipMalloc((void **)&image_d,images*sizeof(eigen_type)), 
		errormsg);
	sprintf(errormsg, "Failed to allocate the recognized index prior to algorithm!");
	cudasafe(
		hipMalloc((void **)&recognized_index_d,sizeof(int)), 
		errormsg);
	sprintf(errormsg, "Failed to allocate for test image pixels!");
	cudasafe(
		hipMalloc((void **)&test_image_d, sizeof(Pixel)*imgsize), 
		errormsg);
	sprintf(errormsg, "Failed to allocate for normalized test image!");
	cudasafe(
		hipMalloc((void **)&test_image_norm, sizeof(eigen_type)*imgsize), 
		errormsg);
	sprintf(errormsg, "Failed to allocate for test image vector 2!");
	cudasafe(
		hipMalloc((void **)&test_image_d2, sizeof(eigen_type) * facessize * (imgsize/256 + 1)), 
		errormsg);
	sprintf(errormsg, "Failed to allocate for mean vector!");
	cudasafe(
		hipMalloc((void **)&mean_d, sizeof(eigen_type) * imgsize), 
		errormsg);
	sprintf(errormsg, "Failed to allocate host->device for eigenfacesT_d!");
	cudasafe(
		hipMalloc((void **)&eigenfacesT_d, sizeof(eigen_type) * imgsize * facessize), 
		errormsg);

	/*  set values to 0 */
	hipMemset(database_d, 0, images*facessize*sizeof(eigen_type));
	hipMemset(image_d, 0, images*sizeof(eigen_type));
	hipMemset(test_image_d, 0, sizeof(Pixel)*imgsize);
	hipMemset(test_image_d2, 0, sizeof(eigen_type) * facessize);

	/*  copy the mean vector to the device  */
	sprintf(errormsg, "Failed to copy host->device for mean vector!");
	cudasafe(
		hipMemcpy(mean_d,mean,sizeof(eigen_type) * imgsize,hipMemcpyHostToDevice), errormsg);
	sprintf(errormsg, "Failed to copy host->device for image database!");
	cudasafe(
		hipMemcpy(eigenfacesT_d, eigenfacesT, sizeof(eigen_type) * imgsize * facessize, hipMemcpyHostToDevice), errormsg);
	
	printf("Cuda Memory Setup Time: %f\n", itime + (double)(clock())/CLOCKS_PER_SEC);
	itime = -(double)(clock())/CLOCKS_PER_SEC;
	
	/*  compare each test image to the database */
	for(i = 0; i<10; i++) {
	    sprintf(inputimage, "ORL_200/%d.ppm", (4*(i+1)));
//		sprintf(inputimage, "../Image/Train/FERET/%d.ppm", (4*(i+1)));
	    Recognition(inputimage, &mean_d, &projectedimages, &eigenfacesT_d, &projectedtrainimages, &images, &imgsize, &facessize, &database_d, &image_d, &recognized_index_d, &test_image_d, &test_image_d2, &test_image_norm);    
	}
	
	printf("Image comparison time: %f\n", itime + (double)(clock())/CLOCKS_PER_SEC);
	itime = -(double)(clock())/CLOCKS_PER_SEC;
	
	/*  Shutdown    */
	cublasShutdown();
	printf("PCA done...\n");
	
	/*  free host memory    */
	free(projectedimages);
	free(eigenfacesT);
	free(projectedtrainimages);
	
	/*  free CUDA memory    */
	hipFree(image_d);
	hipFree(database_d);
	hipFree(recognized_index_d);	
	hipFree(test_image_d);
	hipFree(test_image_norm);
	hipFree(test_image_d2);
	hipFree(mean_d);
	hipFree(eigenfacesT_d);	
	
	printf("Cleanup time: %f\n", itime + (double)(clock())/CLOCKS_PER_SEC);
	printf("Total time: %f\n", totalTime + (double)(clock())/CLOCKS_PER_SEC);
       	
	return(0);
}
